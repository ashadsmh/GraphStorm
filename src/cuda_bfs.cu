#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
using namespace std;

#define THREADS_PER_BLOCK 256

/**
 * CUDA Kernel for BFS frontier expansion
 */
__global__ void bfsKernel(int *rowPtr, int *colIdx, int *frontier, int *nextFrontier, int *visited, int *done, int numNodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numNodes && frontier[tid]) {
        frontier[tid] = 0;
        for (int i = rowPtr[tid]; i < rowPtr[tid + 1]; i++) {
            int neighbor = colIdx[i];
            if (!visited[neighbor]) {
                visited[neighbor] = 1;
                nextFrontier[neighbor] = 1;
                *done = 0;
            }
        }
    }
}

/**
 * Host BFS using CSR graph
 */
void bfsCUDA(vector<int> &rowPtr, vector<int> &colIdx, int start, int numNodes) {
    int *d_rowPtr, *d_colIdx, *d_frontier, *d_nextFrontier, *d_visited, *d_done;
    vector<int> visited(numNodes, 0), frontier(numNodes, 0), nextFrontier(numNodes, 0);

    hipMalloc(&d_rowPtr, rowPtr.size() * sizeof(int));
    hipMalloc(&d_colIdx, colIdx.size() * sizeof(int));
    hipMalloc(&d_frontier, numNodes * sizeof(int));
    hipMalloc(&d_nextFrontier, numNodes * sizeof(int));
    hipMalloc(&d_visited, numNodes * sizeof(int));
    hipMalloc(&d_done, sizeof(int));

    hipMemcpy(d_rowPtr, rowPtr.data(), rowPtr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, colIdx.data(), colIdx.size() * sizeof(int), hipMemcpyHostToDevice);

    visited[start] = 1;
    frontier[start] = 1;
    hipMemcpy(d_visited, visited.data(), numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier, frontier.data(), numNodes * sizeof(int), hipMemcpyHostToDevice);

    cout << "Running CUDA BFS...\n";
    bool finished = false;
    while (!finished) {
        finished = true;
        hipMemcpy(d_done, &finished, sizeof(int), hipMemcpyHostToDevice);
        bfsKernel<<<(numNodes + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(
            d_rowPtr, d_colIdx, d_frontier, d_nextFrontier, d_visited, d_done, numNodes
        );
        hipMemcpy(&finished, d_done, sizeof(int), hipMemcpyDeviceToHost);
    }

    cout << "CUDA BFS traversal complete.\n";

    hipFree(d_rowPtr);
    hipFree(d_colIdx);
    hipFree(d_frontier);
    hipFree(d_nextFrontier);
    hipFree(d_visited);
    hipFree(d_done);
}
